#include "hip/hip_runtime.h"
#include <string>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "md5.hpp"

#define PASSWORDS_PER_KERNEL 4
#define MAX_PASSWORD_LEN 256
#define DIGEST_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void crackMD5(unsigned char* hash_in, char* pass_set, uint32_t len, char* pass_out) {
    // TODO
	unsigned char hash_in_cache[DIGEST_SIZE];
	memcpy(hash_in_cache, hash_in, DIGEST_SIZE);

	// TODO CREATE LOOP WHER IF IS
    for(int id = threadIdx.x + blockIdx.x*blockDim.x ; id < len ; id += gridDim.x*blockDim.x) {
		// Init varibles for password test
        char * pass_test = pass_set + MAX_PASSWORD_LEN * id;
        char pass_cache[MAX_PASSWORD_LEN];
        int pass_len = 0;

		printf("Thread%d: %s\n", threadIdx.x, pass_set+id*MAX_PASSWORD_LEN);
        // Copy and find the length of the password to test
        while(pass_test[pass_len]) {
            pass_cache[pass_len] = pass_test[pass_len];
            ++pass_len;
        }
        pass_cache[pass_len] = 0;
       
	   	// Create hash for password to test
		MD5 md5(pass_cache, pass_len);
              
		// Retrieve created hash
        unsigned char result[DIGEST_SIZE]; // 128 bit
        md5.get_digest(result); // load the result
    
		// Test created hash against hash to crack
		int success = 1;
		for(int i = 0 ; i < DIGEST_SIZE ; ++i ) {
			if(result[i] != hash_in_cache[i]) {	
				success = 0;	
				break;
			}
		}

		// If crack is successful, return result
		if(success) {
			memcpy(pass_out, pass_cache, DIGEST_SIZE);
    	}
	}
}

int main(int argc, char const ** argv) {
    // TODO load a file and stuff

    std::string hash = "5f4dcc3b5aa765d61d8327deb882cf99"; // 'password'

    std::cout << hash << std::endl;

	unsigned char hash_in[17];
	strcpy( (char*) hash_in, hexencode(hash.c_str() ).c_str());

	// device declerations
    char * d_pass_out;
    unsigned char * d_hash_in;
    char * d_passwords;

	// device memory allocations
    gpuErrchk(hipMalloc((void**) &d_pass_out, MAX_PASSWORD_LEN));
    gpuErrchk(hipMalloc((void**) &d_hash_in, 16));
    gpuErrchk(hipMalloc((void**) &d_passwords, PASSWORDS_PER_KERNEL * MAX_PASSWORD_LEN));

	//TODO load all passwords with padding, passwords size = PASSWORDS_PER_KERNEL*MAX_PASSWORD_LEN
    const char * dictionary[] = {"kattj'vel", "passwor", "password", "passwords"};
	char passwords[PASSWORDS_PER_KERNEL * MAX_PASSWORD_LEN] = {0};
	for(int p = 0 ; p < PASSWORDS_PER_KERNEL ; ++p) {
		strcpy(passwords+p*MAX_PASSWORD_LEN, dictionary[p]);	
	}


	// device variable initializing
    gpuErrchk(hipMemcpy(d_hash_in, hash_in, 16, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_passwords, passwords, PASSWORDS_PER_KERNEL*MAX_PASSWORD_LEN , hipMemcpyHostToDevice));
	gpuErrchk(hipMemset(d_pass_out, 0, MAX_PASSWORD_LEN));

	// run crack
	dim3 grid_dim(1,0,0);
	dim3 block_dim(1,0,0); // TODO fix number of threads, make grid dynamic
    crackMD5<<<3,3>>>(d_hash_in, d_passwords, PASSWORDS_PER_KERNEL, d_pass_out);
   
	hipError_t err = hipGetLastError();
	if(err != hipSuccess) {
		printf("ERROR: %s\n", err);
    }


	// retrieve result
    unsigned char result[MAX_PASSWORD_LEN] = {0};
    hipMemcpy(result, d_pass_out, MAX_PASSWORD_LEN, hipMemcpyDeviceToHost);
  

  	// free device memory
	gpuErrchk(hipFree(d_pass_out));
	gpuErrchk(hipFree(d_hash_in));
	gpuErrchk(hipFree(d_passwords));

	// TODO test if there's a result


	// print result
    std::cout << hexdigest(hash_in) << std::endl;
    std::cout << "Password is: " << result << std::endl; 
    return 0;
}
