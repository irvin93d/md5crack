#include "hip/hip_runtime.h"
#include <string>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "md5.hpp"

#define NO_OF_PASSWORDS 4
#define MAX_PASSWORD_LEN 256
#define DIGEST_SIZE 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void crackMD5(unsigned char* hash_in, char* pass_set, uint32_t len, char* pass_out) {
    // TODO
    int id = threadIdx.x;
	unsigned char hash_in_cache[DIGEST_SIZE];
	memcpy(hash_in_cache, hash_in, DIGEST_SIZE);

    if(id == 0) {
		// Init varibles for password test
        char * pass_test = pass_set + MAX_PASSWORD_LEN * id;
        char pass_cache[MAX_PASSWORD_LEN];
        int pass_len = 0;

        // Copy and find the length of the password to test
        while(pass_test[pass_len]) {
            pass_cache[pass_len] = pass_test[pass_len];
            ++pass_len;
        }
        pass_cache[pass_len] = 0;
       
	   	// Create hash for password to test
		MD5 md5(pass_cache, pass_len);
              
		// Retrieve created hash
        unsigned char result[DIGEST_SIZE]; // 128 bit
        md5.get_digest(result); // load the result
    
		// Test created hash against hash to crack
		int success = 1;
		for(int i = 0 ; i < DIGEST_SIZE ; ++i ) {
			if(result[i] != hash_in_cache[i]) {	
				success = 0;	
				break;
			}
		}

		// If crack is successful, return result
		if(success) {
			memcpy(pass_out, "success", DIGEST_SIZE);
    	}
	}
}

int main(int argc, char const ** argv) {
    // TODO load a file and stuff
    std::string hash = "5f4dcc3b5aa765d61d8327deb882cf99"; // 'password'
    char passwords[] = {"password"};

    std::cout << hash << std::endl;

	unsigned char hash_in[17];
	strcpy( (char*) hash_in, hexencode(hash.c_str() ).c_str());

	// device declerations
    char * d_pass_out;
    unsigned char * d_hash_in;
    char * d_passwords;

	// device memory allocations
    gpuErrchk(hipMalloc((void**) &d_pass_out, MAX_PASSWORD_LEN));
    gpuErrchk(hipMalloc((void**) &d_hash_in, 16));
    gpuErrchk(hipMalloc((void**) &d_passwords, MAX_PASSWORD_LEN)); // TODO multiply by number of passwords

	// device variable initializing
    gpuErrchk(hipMemcpy(d_hash_in, hash_in, 16, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_passwords, passwords, MAX_PASSWORD_LEN , hipMemcpyHostToDevice)); // TODO muliply by number of passwords
	gpuErrchk(hipMemset(d_pass_out, 0, MAX_PASSWORD_LEN));

	// run crack
    crackMD5<<<1,1>>>(d_hash_in, d_passwords, 1, d_pass_out);
    
	// retrieve result
    unsigned char result[MAX_PASSWORD_LEN] = {0};
    hipMemcpy(result, d_pass_out, MAX_PASSWORD_LEN, hipMemcpyDeviceToHost);
  

  	// free device memory
	gpuErrchk(hipFree(d_pass_out));
	gpuErrchk(hipFree(d_hash_in));
	gpuErrchk(hipFree(d_passwords));

	// TODO test if there's a result


	// print result
    std::cout << hexdigest(hash_in) << std::endl;
    std::cout << result << std::endl; 
    return 0;
}
