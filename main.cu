#include "hip/hip_runtime.h"
#include <string>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "md5.hpp"
#include <fstream>
#include <errno.h>

#define PASSWORDS_PER_KERNEL 20480
#define MAX_PASSWORD_LEN 256
#define DIGEST_SIZE 16
#define BLOCK_DIM 256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ bool password_found = false;

__global__ void crackMD5(unsigned char* hash_in, char* pass_set, uint32_t len, char* pass_out) {
	unsigned char hash_in_cache[DIGEST_SIZE];
	memcpy(hash_in_cache, hash_in, DIGEST_SIZE);

    for(int id = threadIdx.x + blockIdx.x*blockDim.x ; id < len && !password_found ; id += gridDim.x*blockDim.x) {
		// Init varibles for password test
        char * pass_test = pass_set + MAX_PASSWORD_LEN * id;
        char pass_cache[MAX_PASSWORD_LEN];
        int pass_len = 0;

        // Copy and find the length of the password to test
        while(pass_test[pass_len]) {
            pass_cache[pass_len] = pass_test[pass_len];
            ++pass_len;
        }
        pass_cache[pass_len] = 0;
       
	   	// Create hash for password to test
		MD5 md5(pass_cache, pass_len);
              
		// Retrieve created hash
        unsigned char result[DIGEST_SIZE]; // 128 bit
        md5.get_digest(result); // load the result
    
		// Test created hash against hash to crack
		int success = 1;
		for(int i = 0 ; i < DIGEST_SIZE ; ++i ) {
			if(result[i] != hash_in_cache[i]) {	
				success = 0;	
				break;
			}
		}

		if(password_found)
			break;
		// If crack is successful, return result
		if(success) {
			password_found = true;
			memcpy(pass_out, pass_cache, DIGEST_SIZE);
    	}
	}
}

int main(int argc, char const ** argv) {

	

    char const * hash  = argv[1]; // 'password'

	std::ifstream file("crackstation-human-only.txt");
	if(!file) {
		std::cerr << "Error: " << strerror(errno) << std::endl;
		return(-1);
	}

    std::cout << "Hash: " << hash << std::endl;
    
	unsigned char result[MAX_PASSWORD_LEN] = {0};
	// Convert the hex representation of the hash
    unsigned char hash_in[17];
	strcpy( (char*) hash_in, hexencode(hash).c_str());

	// device declerations
    char * d_pass_out;
    unsigned char * d_hash_in;
    char * d_passwords;

	// device memory allocations
    gpuErrchk(hipMalloc((void**) &d_pass_out, MAX_PASSWORD_LEN));
    gpuErrchk(hipMalloc((void**) &d_hash_in, 16));
    gpuErrchk(hipMalloc((void**) &d_passwords, PASSWORDS_PER_KERNEL * MAX_PASSWORD_LEN));

	int password_found = 0; 
	while(!password_found) {

		//load a chunk of passwords. Passwords are null terminated
		char passwords[PASSWORDS_PER_KERNEL * MAX_PASSWORD_LEN] = {0};
		std::string str;
        for(int p = 0 ; p < PASSWORDS_PER_KERNEL ; ++p) {
            if(!std::getline(file, str)) {
                password_found = -1;
                break;
            }

			strcpy(passwords+p*MAX_PASSWORD_LEN, str.c_str()); // load file row to padded password list
			passwords[p*MAX_PASSWORD_LEN + str.length()-1] = 0; // exchange last character '\n' to '\0'
		}

		// device variable initializing
		gpuErrchk(hipMemcpy(d_hash_in, hash_in, 16, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_passwords, passwords, PASSWORDS_PER_KERNEL*MAX_PASSWORD_LEN , hipMemcpyHostToDevice));
		gpuErrchk(hipMemset(d_pass_out, 0, MAX_PASSWORD_LEN));

		// run crack
		crackMD5<<<(PASSWORDS_PER_KERNEL+BLOCK_DIM-1)/BLOCK_DIM,BLOCK_DIM>>>(d_hash_in, d_passwords, PASSWORDS_PER_KERNEL, d_pass_out);
		hipError_t err = hipGetLastError();
		if(err != hipSuccess) {
			printf("ERROR: %s\n", err);
		}

		// retrieve result
		hipMemcpy(result, d_pass_out, MAX_PASSWORD_LEN, hipMemcpyDeviceToHost);

		if(result[0]){
			password_found = 1;		
		}
	}

	// free device memory
	gpuErrchk(hipFree(d_pass_out));
	gpuErrchk(hipFree(d_hash_in));
	gpuErrchk(hipFree(d_passwords));

	// print result
	if(password_found == 1)
    	std::cout << "Password is: " << result << std::endl; 
   	else
    	std::cout << "Password not found" << std::endl; 

	return 0;
}
